#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#define PI 3.14159265358979323846


#include <stdio.h>

#include <math.h>
#include <omp.h>
#include <ctime>


__global__ void dft2d_kernel(float* in_array, hipComplex* out_array, int width, int height) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height) {
        hipComplex sum = make_hipComplex(0.0f, 0.0f);
        for (int i = 0; i < width; i++) {
            for (int j = 0; j < height; j++) {
                float angle = 2.0f * M_PI * (float)(i * x + j * y) / (float)(width * height);
                hipComplex factor = make_hipComplex(cosf(angle), -sinf(angle));
                hipComplex input_value = make_hipComplex(in_array[j * width + i], 0.0f);
                hipComplex product = hipCmulf(input_value, factor);
                sum = hipCaddf(sum, product);
            }
        }
        out_array[y * width + x] = sum;
    }
}
