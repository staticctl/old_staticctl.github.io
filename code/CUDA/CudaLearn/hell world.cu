﻿
#include "hip/hip_runtime.h"

#include <stdio.h>


/*

__global__ void hello_world(void)
{
    printf("GPU: Hello world!\n");
}
int main(int argc, char** argv)
{
    printf("CPU: Hello world!\n");
    hello_world <<<1, 10 >>> ();
    cudaDeviceReset();//if no this line ,it can not output hello world from gpu
    return 0;
}
*/
/*
* https://github.com/Tony-Tan/CUDA_Freshman
* 3_sum_arrays
*/

#include <stdio.h>
//#include "freshman.h"

/*
void sumArrays(float* a, float* b, float* res, const int size)
{
    for (int i = 0; i < size; i += 4)
    {
        res[i] = a[i] + b[i];
        res[i + 1] = a[i + 1] + b[i + 1];
        res[i + 2] = a[i + 2] + b[i + 2];
        res[i + 3] = a[i + 3] + b[i + 3];
    }

__global__ void sumArraysGPU(float* a, float* b, float* res)
{
    int i = threadIdx.x;
    res[i] = a[i] + b[i];
}
*/
/*
int main(int argc, char** argv)
{
    int dev = 0;
    cudaSetDevice(dev);

    int nElem = 32;
    printf("Vector size:%d\n", nElem);
    int nByte = sizeof(float) * nElem;
    float* a_h = (float*)malloc(nByte);
    float* b_h = (float*)malloc(nByte);
    float* res_h = (float*)malloc(nByte);
    float* res_from_gpu_h = (float*)malloc(nByte);
    memset(res_h, 0, nByte);
    memset(res_from_gpu_h, 0, nByte);

    float* a_d, * b_d, * res_d;
    CHECK(cudaMalloc((float**)&a_d, nByte));
    CHECK(cudaMalloc((float**)&b_d, nByte));
    CHECK(cudaMalloc((float**)&res_d, nByte));

    // 产生浮点型随机数 initialData
    initialData(a_h, nElem);
    initialData(b_h, nElem);

    CHECK(cudaMemcpy(a_d, a_h, nByte, cudaMemcpyHostToDevice));
    CHECK(cudaMemcpy(b_d, b_h, nByte, cudaMemcpyHostToDevice));

    dim3 block(nElem);
    dim3 grid(nElem / block.x);
    sumArraysGPU << <grid, block >> > (a_d, b_d, res_d);
    printf("Execution configuration<<<%d,%d>>>\n", block.x, grid.x);

    CHECK(cudaMemcpy(res_from_gpu_h, res_d, nByte, cudaMemcpyDeviceToHost));
    sumArrays(a_h, b_h, res_h, nElem);

    checkResult(res_h, res_from_gpu_h, nElem);
    cudaFree(a_d);
    cudaFree(b_d);
    cudaFree(res_d);

    free(a_h);
    free(b_h);
    free(res_h);
    free(res_from_gpu_h);

    return 0;
}
*/

/*
*1_check_dimension
*/
/*
#include <cuda_runtime.h>
#include <stdio.h>
__global__ void checkIndex(void)
{
    printf("threadIdx:(%d,%d,%d) blockIdx:(%d,%d,%d) blockDim:(%d,%d,%d)\
  gridDim(%d,%d,%d)\n", threadIdx.x, threadIdx.y, threadIdx.z,
        blockIdx.x, blockIdx.y, blockIdx.z, blockDim.x, blockDim.y, blockDim.z,
        gridDim.x, gridDim.y, gridDim.z);
}
int main(int argc, char** argv)
{
    int nElem = 6;
    dim3 block(3);
    dim3 grid((nElem + block.x - 1) / block.x);
    printf("grid.x %d grid.y %d grid.z %d\n", grid.x, grid.y, grid.z);
    printf("block.x %d block.y %d block.z %d\n", block.x, block.y, block.z);
    checkIndex << <grid, block >> > ();
    cudaDeviceReset();
    return 0;
}
*/


/*
*2_grid_block
*/

#include <stdio.h>
int main(int argc, char** argv)
{
    int nElem = 1024;
    dim3 block(1024);
    dim3 grid((nElem - 1) / block.x + 1);
    printf("grid.x %d block.x %d\n", grid.x, block.x);

    block.x = 512;
    grid.x = (nElem - 1) / block.x + 1;
    printf("grid.x %d block.x %d\n", grid.x, block.x);

    block.x = 256;
    grid.x = (nElem - 1) / block.x + 1;
    printf("grid.x %d block.x %d\n", grid.x, block.x);

    block.x = 128;
    grid.x = (nElem - 1) / block.x + 1;
    printf("grid.x %d block.x %d\n", grid.x, block.x);

    hipDeviceReset();
    return 0;
}